#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include "matrixOperations.cuh"
#include <cmath>

using namespace std;
// Vector Addition
__global__ void addVectors(float* a, float* b, float *c, int size){
	int i = threadIdx.x;
	if (i < size)
		c[i] = a[i] + b[i];
}

__global__ void dotProduct(float* a, float* b, float *c, int size){
	int i = threadIdx.x;
	if (i < size)
		c[i] = a[i] + b[i];
}

// Matrix Addition
__global__ void addMatrices(float* a, float *b, float *c, int n, int m){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (j < n && i < m) {
        c[i * n + j]= a[i * n + j] + b[i * n + j];
    }
}

// Matrix Scaling
__global__ void scaleMatrix(float* a, float* b, float scalar, int n, int m){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (j < n && i < m) {
        b[i * n + j]= a[i * n + j] * scalar;
    }
}



// Matrix Multiplication
__global__ void multiplyMatrices(float *d_A, float *d_B, float *d_C, int M, int N, int P) {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        if(row < N && col < P) {
                float sum = 0.0f;

                // compute the dot product for each row of A and col of B
                for(int i = 0; i < M; ++i) {
                        sum += d_A[row * M + i] * d_B[i * P + col];
                }
                d_C[row * P + col] = sum;
        }
}


// Sum of elements in matrix
__global__ void sumMatrix(float* a, float aSum, int M, int N){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < M && col < N) {
		aSum += a[row * N + col];
	}
}

__global__ void multiplyMatrixElements(float *a, float *b, float *c, int n, int m) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (j < n && i < m) {
        c[i * n + j]= a[i * n + j] + b[i * n + j];
    }
}

namespace MatrixOperations {
	void vector_addition(float* a, float* b, float* c, int n){		
		addVectors<<<1, n>>>(a, b, c, n);
		hipDeviceSynchronize();
	} 

	void matrix_addition(float* a, float* b, float* c, int n, int m) {
		dim3 blockSize(16, 16); 
		dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);
		addMatrices<<<gridSize, blockSize>>>(a, b, c, n, m);
		hipDeviceSynchronize();
	}

	void matrix_scaling(float* a, float* b, float scalar, int n, int m) {
		dim3 blockSize(16, 16);
		dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);
		scaleMatrix<<<gridSize, blockSize>>>(a, b, scalar, n, m);
		hipDeviceSynchronize();

	}

	void matrix_multiplication(float* a, float *b, float* c, int n, int m, int p){
		dim3 blockSize(16, 16);
		dim3 gridDim((p+blockSize.x - 1)/blockSize.x, (m + blockSize.y-1)/blockSize.y);
		multiplyMatrices<<<gridDim, blockSize>>>(a, b, c, m, n, p);
		hipDeviceSynchronize();
	}

	void multiply_matrix_elements(float* a, float* b, float* c, int n, int m) {
		dim3 blockSize(16, 16); 
		dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);
		multiplyMatrixElements<<<gridSize, blockSize>>>(a, b, c, n, m);
		hipDeviceSynchronize();
	}
}
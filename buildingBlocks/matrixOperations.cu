#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include "matrixOperations.cuh"
#include <cmath>

using namespace std;
// Vector Addition
__global__ void addVectors(float* a, float* b, float *c, int size){
	int i = threadIdx.x;
	if (i < size)
		c[i] = a[i] + b[i];
}


// Matrix Addition
__global__ void addMatrices(float* a, float *b, float *c, int n, int m){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (j < m && i < n) {
        c[i * n + j]= a[i * n + j] + b[i * n + j];
    }
}

// Matrix Scaling
__global__ void scaleMatrix(float* a, float* b, float scalar, int n, int m){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (j < m && i < n) {
        b[i * n + j]= a[i * n + j] * scalar;
    }
}



// Matrix Multiplication
// Try Strassen Algorithm????
// #define TILE_WIDTH 16

// __global__ void multiplyMatrices(float *d_A, float *d_B, float *d_C, int M, int N, int P) {
//     __shared__ float tile_A[TILE_WIDTH][TILE_WIDTH];
//     __shared__ float tile_B[TILE_WIDTH][TILE_WIDTH];

//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     int col = blockIdx.x * blockDim.x + threadIdx.x;
//     float sum = 0.0f;

//     for (int k = 0; k < (N + TILE_WIDTH - 1) / TILE_WIDTH; ++k) {
//         if (row < M && k*TILE_WIDTH + threadIdx.x < N)
//             tile_A[threadIdx.y][threadIdx.x] = d_A[row*N + k*TILE_WIDTH + threadIdx.x];
//         else
//             tile_A[threadIdx.y][threadIdx.x] = 0.0;

//         if (col < P && k*TILE_WIDTH + threadIdx.y < N)
//             tile_B[threadIdx.y][threadIdx.x] = d_B[(k*TILE_WIDTH + threadIdx.y)*P + col];
//         else
//             tile_B[threadIdx.y][threadIdx.x] = 0.0;

//         __syncthreads();

//         for (int n = 0; n < TILE_WIDTH; ++n) {
//             sum += tile_A[threadIdx.y][n] * tile_B[n][threadIdx.x];
//         }
//         __syncthreads();
//     }

//     if (row < M && col < P) {
//         d_C[row*P + col] = sum;
//     }
// }
__global__ void multiplyMatrices(float *d_A, float *d_B, float *d_C, int M, int N, int P) {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        if(row < M && col < P) {
                float sum = 0.0f;

                // compute the dot product for each row of A and col of B
                for(int i = 0; i < N; ++i) {
                        sum += d_A[row * N + i] * d_B[i * P + col];
                }
                d_C[row * P + col] = sum;
        }
}


// Sum of elements in matrix
__global__ void sumMatrix(float* a, float aSum, int M, int N){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < M && col < N) {
		aSum += a[row * N + col];
	}
}


namespace MatrixOperations {
	void vector_addition(float* a, float* b, float* c, int n){		
		addVectors<<<1, n>>>(a, b, c, n);
		hipDeviceSynchronize();
	} 

	void matrix_addition(float* a, float* b, float* c, int n, int m) {
		dim3 blockSize(16, 16); 
		dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);
		addMatrices<<<gridSize, blockSize>>>(a, b, c, n, m);
		hipDeviceSynchronize();
	}

	void matrix_scaling(float* a, float* b, float scalar, int n, int m) {
		dim3 blockSize(16, 16);
		dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);
		scaleMatrix<<<gridSize, blockSize>>>(a, b, scalar, n, m);
		hipDeviceSynchronize();

	}

	void matrix_multiplication(float* a, float *b, float* c, int n, int m, int p){
		dim3 blockSize(16, 16);
		dim3 gridDim((p+blockSize.x - 1)/blockSize.x, (m + blockSize.y-1)/blockSize.y);
		multiplyMatrices<<<gridDim, blockSize>>>(a, b, c, m, n, p);
		hipDeviceSynchronize();
	}

}
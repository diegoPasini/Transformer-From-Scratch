#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <vector>
#include <random>
#include <cstdlib>
#include <cmath>
#include "../Tensor.cuh"
#include "layer.h"

#include <memory>

using namespace std;


// Implementation of a linear layer WITH ADAM OPTIMIZER
class LinearLayer : public Layer {
	private:
		int input_features;
		int output_features;
		float learning_rate;
		// Number of rows equal to the number of neurons in the previous layer.
		// Number of columns equal to the number of neurons in the next layer.
		unique_ptr<Tensor> weights;
    	unique_ptr<Tensor> bias;
		Tensor inputs;
		Tensor outputs;
		unique_ptr<Tensor> V_dw;
		unique_ptr<Tensor> V_db;
		unique_ptr<Tensor> S_dw;
		unique_ptr<Tensor> S_db;

		float beta_1 = 0.9;
		float beta_2 = 0.999;
		float epsilon = 10^(-8);
		
		void initialize_weights() {
			float k = 1 / float(input_features); 
			random_device rd;  // a seed source for the random number engine
    		mt19937 gen(rd());
			// First we initialize the weights based on a uniform distribution 
			// X ~ U(-√(k), √(k))
			uniform_real_distribution<float> distr(-sqrt(k), sqrt(k));
			vector<float> weightsTemp(output_features * input_features);
			for (int i = 0; i < output_features * input_features; i++) {
				float generated = distr(gen);
				weightsTemp[i] = generated;
			}
			
			vector<int> dims = {output_features, input_features};
			weights = make_unique<Tensor>(weightsTemp, dims, "cuda");
			vector<float> biasTemp(output_features);

			for (int i = 0; i < output_features; i++) {
				biasTemp[i] = distr(gen);
			}
			vector<int> dim = {output_features, 1};
			bias = make_unique<Tensor>(biasTemp, dim, "cuda");
		}


	public: 
		LinearLayer(int input_features, int output_features, float learning_rate) 
		: learning_rate(learning_rate) { 
			this->input_features = input_features;
			this->output_features = output_features;
			initialize_weights();
		}

		// Destructor
		~LinearLayer() {
		}


		Tensor forward(Tensor x) {
			this->inputs = x;
			x = (*weights * x);
			x = x + *bias;
			return x;
		}

		Tensor backward(Tensor gammaPrev) {
			inputs.transpose();
			Tensor newGamma = gammaPrev * inputs;
			Tensor dW = -1.0f * newGamma;

			Tensor db;
			if (dW.getTotalValues() != 1) {
				vector<float> biasValues(dW.getTotalValues() / input_features);
				for (int i = 0; i < output_features; i++) {
					for (int j = 0; j < input_features; j++) {
						biasValues[i] += x[{i, j}]; 
					}
				}
				db = Tensor(biasValues, {output_features, 1}, "cuda"); 

			} else {
				db = dW;
				*bias = *bias + (dW);
			}

			// Mometum
			if (V_dw) {
				*V_dw = ((beta_1) * *V_dw) + ((1 - beta_1) * dW);
				*V_db = ((beta_1) * *V_db) + ((1 - beta_1) * db);
				*S_dw = ((beta_2) * *S_dw) + ((1 - beta_2) * multiply(dW, dW));
				*S_db = ((beta_2) * *S_db) + ((1 - beta_2) * multiply(db, db));
				

			} else {
				*V_dw = 
				*V_db = 
				*S_dw = 
				*S_db = 
			}

			// RMS Prop 



			*weights = *weights + *V_dw; 
			*bias = *bias + (d_bias);

			//cout << "New Biases: " << (*bias).toString() << endl;
			//cout << "Return Weights Device: " << (*weights).getDevice() << endl;

			Tensor returnWeights = *weights;
			//cout << "Return Weights Device: " << returnWeights.getDevice() << endl;

			returnWeights.transpose();
			//cout << "Return Weights Device: " << returnWeights.getDevice() << endl;
			//cout << "Return GamaPrev Device: " << gammaPrev.getDevice() << endl;

			Tensor outputGradient = returnWeights * gammaPrev;
			
			//cout << "Output Gradient: " << (outputGradient).toString() << endl;
			inputs.transpose();
			return outputGradient;
		}

		string toStringWeights() {
			return (*weights).toString();
		}

		string toStringBiases() {
			return (*bias).toString();
		}


};

// // linear layer cuda implementation
// __global__ void linear_layer_forward(float* x, float* w, float* b, float* c, float alpha, int size){
// 	int i = blockDim.x * blockIdx.x + threadIdx.x;exit
// 	if (i < size)
// 		c[i] = w[i] * x[i] + b[i];
// }
	
// __global__ void linear_layer_backward(float){
// 	return ;
// }


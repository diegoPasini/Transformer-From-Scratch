#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <string>
#include "layer.h"

class ReLU : public Layer {
	public: 
		int input_features;
		int output_features = 1;
		float output;
		ReLU(int input_features) {
            this->input_features = input_features;
        }
        
        ~ReLU() {}

        // NEED CUDA IMPLEMENTATION FOR THIS
        Tensor forward(Tensor x) {
            float sum_of_elems = 0;
            for(int i = 0; i < x.getTotalValues(); i++) {
                sum_of_elems += x.getValues()[i];
            }
            output = fmax(0, sum_of_elems);
            return Tensor({output}, {1}, "cuda");
        }   


        Tensor backward(Tensor gammaPrev) {
            gammaPrev = gammaPrev;
            vector<float> values = gammaPrev.getValues();
            vector<float> newValues(input_features);
            for (int i = 0; i < input_features; i++) {
                newValues[i] = values[0];
            }
            return Tensor(newValues, {1, input_features}, "cuda");;
        }
};
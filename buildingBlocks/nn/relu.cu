
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <cmath>


// relu cuda implementation
__global__ void relu_forward(float* a, float* b, float alpha, int size){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size)
		b[i] = fmaxf(alpha * a[i], b[i]);
}


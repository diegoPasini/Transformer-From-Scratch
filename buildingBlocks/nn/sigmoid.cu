#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <string>
#include "layer.h"
//#include "linearlayer.cu"
#include <cmath>
#include <numeric>
#include "../Tensor.cuh"


class Sigmoid : public Layer {
    public:
        int input_features;
        int output_features = 1;
        float output;
        Sigmoid(int input_features) {
            this->input_features = input_features;
        }
        
        ~Sigmoid() {}

        // NEED CUDA IMPLEMENTATION FOR THIS
        Tensor forward(Tensor x) {
            float sum_of_elems = 0;
            for(int i = 0; i < x.getTotalValues(); i++) {
                sum_of_elems += x.getValues()[i];
            }
            output = 1 / (1 + exp(-sum_of_elems));
            return Tensor({output}, {1}, "cuda");
        }   


        Tensor backward(Tensor gammaPrev) {
            gammaPrev = output * (1-output) * gammaPrev;
            vector<float> values = gammaPrev.getValues();
            vector<float> newValues(input_features);
            for (int i = 0; i < input_features; i++) {
                newValues[i] = values[0];
            }
            return Tensor(newValues, {1, input_features}, "cuda");;
        }
};
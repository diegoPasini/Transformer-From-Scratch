
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void dropout_kernel(float* input, float* output, float dropout_prob, int size, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        float rand_val = hiprand_uniform(&state);
        output[idx] = (rand_val > dropout_prob) ? input[idx] / (1.0f - dropout_prob) : 0.0f;
    }
}

void dropout(float* input, float* output, float dropout_prob, int size) {
    float *d_input, *d_output;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    unsigned long long seed = 1234ULL;
    dropout_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, dropout_prob, size, seed);

    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
